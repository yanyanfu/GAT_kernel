#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include "kernel.h"
#include "invoke.h"
#include <cassert>
#include <iostream>
#include <limits>
#define SIZE 1024
#define Block 1
#define FULL_WARP_MASK 0xFFFFFFFF


template <class T>
__device__ T warp_reduce(T val){
    for(int offset=32/2;offset>0;offset/=2)
        val+= __shfl_down_sync (FULL_WARP_MASK,val,offset);
    return val;

}

typedef float (*op_scalar_fn)(float, float);

__device__ inline float add_scalar(float x, float y) {
    return x + y;
}

__device__ inline float sub_scalar(float x, float y) {
    return x - y;
}

__device__ inline float max_scalar(float x, float y) {
    if(x>y) return x;
    else return y;
}

__device__ inline float min_scalar(float x, float y) {
    if(x<y) return x;
    else return y;
}

__device__ inline float mul_scalar(float x, float y) {
    return x * y;
}

__device__ inline float div_scalar(float x, float y) {
    return x / y;
}

__device__ op_scalar_fn  p_mul = mul_scalar;
__device__ op_scalar_fn  p_div = div_scalar;
__device__ op_scalar_fn  p_add = add_scalar;
__device__ op_scalar_fn  p_sub = sub_scalar;
__device__ op_scalar_fn  p_min = min_scalar;
__device__ op_scalar_fn  p_max = max_scalar;

//to be used if host is sending function pointer to kernel
inline op_scalar_fn get_fn(op_t op) {
    op_scalar_fn op_fn;

    if (op == eDIV) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_div), sizeof(op_scalar_fn));
        //op_fn = div_scalar;
    } else if (op == eSUB) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_sub), sizeof(op_scalar_fn));
        //op_fn = sub_scalar;
    } else if (op == eSUM) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_add), sizeof(op_scalar_fn));
        //op_fn = add_scalar;
    } else if (op == eMUL) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_mul), sizeof(op_scalar_fn));
        //op_fn = mul_scalar;
    } else if (op == eMIN) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_min), sizeof(op_scalar_fn));
        //op_fn = min_scalar;
    } else if (op == eMAX) {
        hipMemcpyFromSymbol(&op_fn, HIP_SYMBOL(p_max), sizeof(op_scalar_fn));
        //op_fn = max_scalar;
    } else {
        assert(0);
    }
    return op_fn;
}

//if the kernel itself need the fuction pointer
__device__ inline op_scalar_fn get_fn_kernel(op_t op) {
    op_scalar_fn op_fn;

    if (op == eDIV) {
        op_fn = div_scalar;
    } else if (op == eSUB) {
        op_fn = sub_scalar;
    } else if (op == eSUM) {
        op_fn = add_scalar;
    } else if (op == eMUL) {
        op_fn = mul_scalar;
    } else if (op == eMIN) {
        op_fn = min_scalar;
    } else if (op == eMAX) {
        op_fn = max_scalar;
    } else {
        assert(0);
    }
    return op_fn;
}

__global__ void spmm(int64_t * s_row, int64_t * s_col, float * s_value, float * d, float * y, const int64_t v_count, const int64_t dim) 
{
    //TODO
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= v_count)
        return;

    float * output = y + dim * row;
    for (int i = 0; i < dim; i++){
        for (int j = s_row[row]; j < s_row[row+1]; j++){
            float * row_d = d + dim * s_col[j];
            output[i] += s_value[j] * row_d [i];
        }
    }
}

void invoke_spmm(int64_t * s_row, int64_t * s_col, float * s_value, float * d, float * y, const int64_t v_count, const int64_t dim)
{
    //int warp_size=32; 
    int block_size = 1024;
    int nBlocks = ceil(v_count/(float)block_size); // TODO 
    //spmm_warp <<<nBlocks,block_size>>> (obj1, x1.data_ptr, y1.data_ptr, op, true, true, dim);
    //spmm <<<nBlocks,block_size>>> (obj1, x1.data_ptr, y1.data_ptr, op, reverse, norm, dim); 
    spmm <<<nBlocks, block_size>>> (s_row, s_col, s_value, d, y, v_count, dim);
    hipDeviceSynchronize();
}

//warp per row (best)
//__global__ void spmm_warp(const csr_t* __restrict__ obj1, float* x, float * y, op_t op, const bool reverse, const bool norm, const int dim)
// {
//     //TODO

// }

// graph_t * invoke_init_graph(vid_t v_count, vid_t dst_size, vid_t * offset_csr, void * nebrs_csr, vid_t * offset_csc, void * nebrs_csc) {

//     //Let us make a cpu graph first
//     graph_t g;
//     g.init_cpu(v_count, dst_size, 
//             offset_csr, nebrs_csr,
//             offset_csc, nebrs_csc);

//     graph_t * graph = (graph_t*) malloc(sizeof(graph_t));
//     hipMallocManaged( & graph->csr,  sizeof(csr_t));

//     vid_t edge_count = offset_csr[v_count];
//     vid_t * offset_csr_gpu;
//     vid_t * offset_csc_gpu;
//     char * nebrs_csr_gpu;
//     char * nebrs_csc_gpu;

//     hipMallocManaged( & offset_csr_gpu, (v_count + 1) * sizeof(vid_t));
//     hipMallocManaged( & nebrs_csr_gpu, edge_count * dst_size);

//     //memcopy
//     hipMemcpy(offset_csr_gpu, offset_csr, (v_count + 1) * sizeof(vid_t), hipMemcpyHostToDevice);
//     hipMemcpy(nebrs_csr_gpu, nebrs_csr, edge_count * dst_size, hipMemcpyHostToDevice);


//     if (nebrs_csr == nebrs_csc) {
//         graph->csc = graph->csr;
//         offset_csc_gpu = offset_csr_gpu;
//         nebrs_csc_gpu = nebrs_csr_gpu;
//     } else {
//         hipMallocManaged( & graph->csc,  sizeof(csr_t));
//         hipMallocManaged( & offset_csc_gpu, (v_count + 1) * sizeof(vid_t));
//         hipMallocManaged( & nebrs_csc_gpu, edge_count * dst_size);

//         hipMemcpy(nebrs_csc_gpu, nebrs_csc, edge_count * dst_size, hipMemcpyHostToDevice);
//         hipMemcpy(offset_csc_gpu, offset_csc, (v_count + 1) * sizeof(vid_t), hipMemcpyHostToDevice);
//     }

//     //printf("invoke init graph called\n");
//     graph -> init(v_count, dst_size, offset_csr_gpu, nebrs_csr_gpu, offset_csc_gpu, nebrs_csc_gpu);

//     return graph;

// }

